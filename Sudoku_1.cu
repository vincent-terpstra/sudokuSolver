#include "hip/hip_runtime.h"
/**
* Vincent Terpstra
* Sudoku.cu
* March 18 / 2019
* An Optimistic approach to solving a Sudoku on a CUDA enabled GPU
*    Assumes that the puzzle is deterministic(single solvable solution)
*        AND each next step can be found with the kernel
* KERNEL: educatedGuess
*   searches each square in a box for
*    squares that have only a single appropiate value
*    OR values that (in the box) can only fit in one square
*/

#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <iomanip>
// CUDA header file
#include "hip/hip_runtime.h"
#include <>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#include <stdio.h> 
// UNASSIGNED is used for empty cells in sudoku grid 
#define UNASSIGNED 0 
// N is used for the size of Sudoku grid. Size will be NxN 
#define BOXWIDTH 3
#define N (BOXWIDTH * BOXWIDTH)

/*
 * kernel to solve a sudoku
 * Input: sudoku puzzle partitioned into boxes
 *	* d_a = the sudoku puzzle
 *	figures out what values can fit in each square
 *  figures out how many spots each value can go
 *  assigns the appropiate values,
 *	saves to addedIdx to show that there is a change
 */

__global__ void educatedGuess(int * d_a, int * addedIdx) {
	int idx = threadIdx.x + BOXWIDTH * threadIdx.y;
	int gridX = threadIdx.x + BOXWIDTH * blockIdx.x;
	int gridY = threadIdx.y + BOXWIDTH * blockIdx.y;
	int gridIdx = gridX + N * gridY;
	__shared__ bool hasValue[N]; //If the value occurs in the box
	__shared__ int  inBox[N];	 //Number of places each integer can go in the box
	hasValue[idx] = false;
	inBox[idx] = 0;
	__syncthreads();
	int at = d_a[gridIdx];
	if (at != 0)
		hasValue[at - 1] = true;
	__syncthreads();
	if (at != 0)
		return;
	//For remembering which values were seen in the rows and columns
	bool foundVal[N];
	for (int i = 0; i < N; ++i)
		foundVal[i] = hasValue[i];

	for (int check = 0; check < N; check++) {
		foundVal[d_a[N * check + gridX] - 1] = true;
		foundVal[d_a[N * gridY + check] - 1] = true;
	}
	int fndVals = 0;
	for (int i = 0; i < N; ++i)
		if (!foundVal[i]) {
			fndVals++;
			at = i + 1;
		}
	if (fndVals == 1) {
		//Only one possible value for this index
		d_a[gridIdx] = at;        //assign value
		addedIdx[0] = gridIdx;   //to tell host that the table has changed
		inBox[at - 1] = 4; //Prevent one index per value
	}
	__syncthreads();
	//Calculate the number of places each integer can go in the box
	for (int i = 0; i < N; ++i) {
		int num = (idx + i) % N; //keep each thread on a seperate idx
		if (!foundVal[num])
			inBox[num]++;
		__syncthreads();
	}
	for (int i = 0; i < N; ++i) {
		//if there is only one possible index for that value assign the value
		if (inBox[i] == 1 && !foundVal[i]) {
			d_a[gridIdx] = i + 1;    //assign value
			addedIdx[0] = gridIdx;   //to tell host that the table has changed
		}
	}
}

/* A utility function to print grid  */
void printGrid(int grid[N][N])
{
	for (int row = 0; row < N; row++) {
		for (int col = 0; col < N; col++)
			printf("%3d", grid[row][col]);
		printf("\n");
	}
}
__global__ void superSolve(int * d_a) {
	__shared__ bool rowHas[N][N];
	__shared__ bool colHas[N][N];
	__shared__ bool boxHas[N][N];
	__shared__ int added, past;
	
	int row = threadIdx.x;
	int col = threadIdx.y;
	int box = row / BOXWIDTH + (col / BOXWIDTH) * BOXWIDTH;
	
	int gridIdx = col * N + row;
	int at = d_a[gridIdx];
	
	if (!gridIdx) { //only 0 needs to set changed
			added = -1;
			past  = -2;
		}
		rowHas[col][row] = false;
		colHas[col][row] = false;
		boxHas[col][row] = false;
	__syncthreads();

	if (at != UNASSIGNED) {
		rowHas[row][at - 1] = true;
		colHas[col][at - 1] = true;
		boxHas[box][at - 1] = true;
	}

	while (added != past) {
		__syncthreads();
		if(!gridIdx)
			past = added;
		if (at == 0) {
			int count = 0;
			for (int num = 0; num < N; ++num) {
				if (!(rowHas[row][num] || colHas[col][num] || boxHas[box][num])) {
					count++;
					at = num + 1;
				}
			}
			if (count == 1) {
				d_a[gridIdx] = at;
				rowHas[row][at - 1] = true;
				colHas[col][at - 1] = true;
				boxHas[box][at - 1] = true;
				added = gridIdx;
			} else {
				at = UNASSIGNED;
			}	
		}
		__syncthreads();
	}
}
/* Driver Program to test above functions */
int main()
{ /* 0 means unassigned cells */
   int grid[N][N] =
   { {3, 0, 6, 5, 0, 8, 4, 0, 0},
   {5, 2, 0, 0, 0, 0, 0, 0, 0},
   {0, 8, 7, 0, 0, 0, 0, 3, 1},
   {0, 0, 3, 0, 1, 0, 0, 8, 0},
   {9, 0, 0, 8, 6, 3, 0, 0, 5},
   {0, 5, 0, 0, 9, 0, 6, 0, 0},
   {1, 3, 0, 0, 0, 0, 2, 5, 0},
   {0, 0, 0, 0, 0, 0, 0, 7, 4},
   {0, 0, 5, 2, 0, 6, 3, 0, 0} };

   /**
   int grid[N][N] =
   {{0,  8,   0,  0,  0,  0,  0,  3,  0,  0,  0, 10,  9,  7, 11, 0},
   {0,  9,  15, 13,  0, 10,  0,  0,  2,  6,  8, 16,  0,  0,  0, 0},
   {0,  0,  16,  0, 15,  0,  8,  0,  9,  0,  0,  0,  6,  0,  2, 0},
   {1,  0,   2,  0,  9, 11,  4,  6, 15,  3,  5,  7,  0,  0, 12, 0},
   {16, 6,   4,  0,  5,  2,  0,  0,  1,  0,  0,  0, 11,  0,  0, 12},
   {5,  11,  0,  0,  0,  3,  0, 15,  0, 16,  0, 13,  0,  1,  0, 8},
   {0,  0,   3,  0,  0,  6, 11, 14,  0,  5,  7,  0,  0,  9,  0, 0},
   {0,  0,   0, 14,  8,  0, 10,  0,  0, 11, 12,  0,  0,  0,  0, 0},
   {0,  7,  13,  0,  0,  0,  0, 12,  0,  8,  9,  0,  0,  0,  3, 0},
   {0,  0,  11,  9,  0,  7,  0,  0,  0,  0,  0, 12,  0,  8, 16, 5},
   {0,  0,  10,  0, 11, 13,  0,  0,  0,  0,  0,  3, 12,  0,  6, 0},
   {0,  5,   0,  0, 10, 15,  0,  1,  7,  2,  0,  0, 14, 11,  0, 0},
   {0,  0,   5,  0,  0, 12, 14,  0,  0, 10,  0,  0, 15,  0,  0, 4},
   {9,  0,  14,  6,  0,  0,  1,  0, 16,  0,  2,  0,  3,  0, 13, 0},
   {8,  13,  0,  4,  0,  0,  0,  0, 12,  7,  3,  0,  0,  6,  0, 0},
   {0,  16, 12,  0,  0,  5,  0,  9,  0, 13, 14,  4,  1,  0,  0, 0} };
   /**

	int grid[N][N] =
	{ {1,  0,   4,  0, 25,  0, 19,  0,  0,  10,  21, 8,  0,  14, 0,  6,  12,   9,  0,  0,  0,  0,  0,  0,  5},{5,  0,  19, 23, 24,  0, 22,  12,  0,  0,  16, 6,  0,  20,  0,  18,  0,   25,  14,  13,  10, 11,  0,  1,  15},{0,  0,   0,  0,  0,  0,  21,  5,  0,  20,  11,  10,  0,  1,  0,  4,  8,   24,  23,  15,  18,  0,  16,  22,  19},

 {0, 7, 21, 8, 18, 0, 0, 0, 11, 0, 5, 0, 0, 24, 0, 0, 0, 17, 22, 1, 9, 6, 25, 0, 0}, {0, 13, 15, 0, 22, 14, 0, 18, 0, 16, 0, 0, 0, 4, 0, 0, 0, 19, 0, 0, 0, 24, 20, 21, 17}, {12, 0, 11, 0, 6, 0, 0, 0, 0, 15, 0, 0, 0, 0, 21, 25, 19, 0, 4, 0, 22, 14, 0, 20, 0}, {8, 0, 0, 21, 0, 16, 0, 0, 0, 2, 0, 3, 0, 0, 0, 0, 17, 23, 18, 22, 0, 0, 0, 24, 6}, {4, 0, 14, 18, 7, 9, 0, 22, 21, 19, 0, 0, 0, 2, 0, 5, 0, 0, 0, 6, 16, 15, 0, 11, 12}, {22, 0, 24, 0, 23, 0, 0, 11, 0, 7, 0, 0, 4, 0, 14, 0, 2, 12, 0, 8, 5, 19, 0, 25, 9}, {20, 0, 0, 0, 5, 0, 0, 0, 0, 17, 9, 0, 12, 18, 0, 1, 0, 0, 7, 24, 0, 0, 0, 13, 4}, {13, 0, 0, 5, 0, 2, 23, 14, 4, 18, 22, 0, 17, 0, 0, 20, 0, 1, 9, 21, 12, 0, 0, 8, 11}, {14, 23, 0, 24, 0, 0, 0, 0, 0, 0, 0, 0, 20, 25, 0, 3, 4, 13, 0, 11, 21, 9, 5, 18, 22}, {7, 0, 0, 11, 17, 20, 24, 0, 0, 0, 3, 4, 1, 12, 0, 0, 6, 14, 0, 5, 25, 13, 0, 0, 0}, {0, 0, 16, 9, 0, 17, 11, 7, 10, 25, 0, 0, 0, 13, 6, 0, 0, 18, 0, 0, 19, 4, 0, 0, 20}, {6, 15, 0, 19, 4, 13, 0, 0, 5, 0, 18, 11, 0, 0, 9, 8, 22, 16, 25, 10, 7, 0, 0, 0, 0}, {0, 0, 0, 2, 0, 0, 10, 19, 3, 0, 1, 0, 22, 9, 4, 11, 15, 0, 20, 0, 0, 8, 23, 0, 25}, {0, 24, 8, 13, 1, 0, 0, 4, 20, 0, 17, 14, 0, 0, 18, 0, 16, 22, 5, 0, 11, 0, 10, 0, 0}, {23, 10, 0, 0, 0, 0, 0, 0, 18, 0, 6, 0, 16, 0, 0, 17, 1, 0, 13, 0, 0, 3, 19, 12, 0}, {25, 5, 0, 14, 11, 0, 17, 0, 8, 24, 13, 0, 19, 23, 15, 9, 0, 0, 12, 0, 20, 0, 22, 0, 7}, {0, 0, 17, 4, 0, 22, 15, 0, 23, 11, 12, 25, 0, 0, 0, 0, 18, 8, 0, 7, 0, 0, 14, 0, 13}, {19, 6, 23, 22, 8, 0, 0, 1, 25, 4, 14, 2, 0, 3, 7, 13, 10, 11, 16, 0, 0, 0, 0, 0, 0}, {0, 4, 0, 17, 0, 3, 0, 24, 0, 8, 20, 23, 11, 10, 25, 22, 0, 0, 0, 12, 13, 2, 18, 6, 0}, {0, 0, 7, 16, 0, 0, 6, 17, 2, 21, 0, 18, 0, 0, 0, 19, 0, 0, 8, 0, 0, 0, 0, 4, 0}, {18, 9, 25, 1, 2, 11, 0, 0, 13, 22, 4, 0, 21, 0, 5, 0, 23, 7, 0, 0, 15, 0, 3, 0, 8}, {0, 21, 10, 0, 0, 12, 0, 20, 16, 0, 19, 0, 0, 0, 0, 15, 14, 4, 2, 18, 23, 25, 11, 7, 0} }; /**/
	/**/
	int* d_a;      //Table
	int* d_result; //Table change indicator
	hipMalloc((void**)&d_a, N*N * sizeof(int));
	hipMalloc((void**)&d_result, sizeof(int));

	//Copy Sudoku over
	hipMemcpy(d_a, grid, N*N * sizeof(int), hipMemcpyHostToDevice);

	//Solve the Sudoku
	dim3 block(N, N);
	superSolve << <1, block >> > (d_a);

	//Copy Sudoku back
	hipMemcpy(grid, d_a, N*N * sizeof(int), hipMemcpyDeviceToHost);
	printGrid(grid);

	hipFree(d_a);
	hipFree(d_result);

	return 0;

}
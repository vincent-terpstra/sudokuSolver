#include "hip/hip_runtime.h"
#include <stdio.h>
// CUDA header file
#include "hip/hip_runtime.h"
#include <>
#ifndef __HIPCC__
#define __HIPCC__
#endif
// UNASSIGNED is used for empty cells in Sudoku grid 
#define UNASSIGNED 0
// BOX_W is used for the length of one of the square sub-regions of the Sudoku grid.
// Overall length will be N * N.
#define BOX_W 5
#define N (BOX_W * BOX_W)


__global__ void solve(int* d_a) {
	// Used to remember which row | col | box ( section ) have which values
	__shared__ bool rowHas[N][N];
	__shared__ bool colHas[N][N];
	__shared__ bool boxHas[N][N];
	// Used to ensure that the table has changed
	__shared__ bool changed;
	
	// Number of spaces which can place the number in each section
	__shared__ char rowCount[N][N];
	__shared__ char colCount[N][N];
	__shared__ char boxCount[N][N];
	// Where the square is located in the Sudoku
	char row = threadIdx.x;
	char col = threadIdx.y;
	int box = row / BOX_W + (col / BOX_W) * BOX_W;

	int gridIdx = col * N + row;
	char at = d_a[gridIdx];
	
	// Unique identifier for each square in row, col, box
	// Corresponds to the generic Sudoku Solve
	// Using a Sudoku to solve a Sudoku !!!
	int offset = col + (row % BOX_W) * BOX_W + (box % BOX_W);
	int m_offset = offset + N;
	// Square's location in the Sudoku

	char count = 0; //Number of values which can fit in this square
	int notSeen = 0; //Boolean Array as an Integer

	if (gridIdx == 0) changed = true;
	rowHas[col][row] = false;
	colHas[col][row] = false;
	boxHas[col][row] = false;

	rowCount[col][row] = 0;
	colCount[col][row] = 0;
	boxCount[col][row] = 0;

	__syncthreads();
	if (at != UNASSIGNED) {
		rowHas[row][at - 1] = true;
		colHas[col][at - 1] = true;
		boxHas[box][at - 1] = true;
	}
	__syncthreads();
	
	int b_shuttle = 1;
	for (int idx = offset; idx < m_offset; ++idx) {
		int num = idx % N;
		if (at == UNASSIGNED && !(rowHas[row][num] || boxHas[box][num] || colHas[col][num])) {
			notSeen |= b_shuttle;	//this value can go here
			++count;				//how many values this square can have

			//how many values this section can have
			rowCount[row][num]++;
			colCount[col][num]++;
			boxCount[box][num]++;
		}
		__syncthreads();
		b_shuttle <<= 1;
	}
	if (at == UNASSIGNED && count == 0) //NOT POSSIBLE SUDOKU
		changed = false;
	__syncthreads();
	
	
	// Previous loop has not changed any values
	
	while(changed){
		__syncthreads();
		bool inSection = true;
		if (gridIdx == 0) // forget previous change
			changed = false;
		int guess = 0; // last value found which can fit in this square
		
		int b_shuttle = 1;
		for (int idx = offset; idx < m_offset; ++idx) {
			// Ensures that every square in each section is working on a different number in the section
			int num = idx % N;
			if (b_shuttle & notSeen) {
				if (rowHas[row][num] || boxHas[box][num] || colHas[col][num]) {
					notSeen ^= b_shuttle;
					--count;
					rowCount[row][num]--;
					colCount[col][num]--;
					boxCount[box][num]--;
				} else if (inSection) {
					guess = num;
				}
			}
			__syncthreads();
			if ((b_shuttle & notSeen) && (rowCount[row][num] == 1 || boxCount[box][num] == 1 || colCount[col][num] == 1))
				inSection = false;

			b_shuttle <<= 1;
		}

		if (count == 1 || !inSection) {
			at = guess + 1;
			notSeen = count = 0;
			rowHas[row][guess] = true;
			colHas[col][guess] = true;
			boxHas[box][guess] = true;
			changed = true;
		}
		__syncthreads();
	};

	if (!(rowHas[row][col] && colHas[row][col] && boxHas[box][col]))
		changed = true; //HAVE NOT SOLVED the sudoku
	if (changed && gridIdx == 0)
		at = 0;
	d_a[gridIdx] = at;
}

void print(int result[N][N]) {
	for (int row = 0; row < N; row++) {
		for (int col = 0; col < N; col++)
			printf("%3d", result[row][col]);
		printf("\n");
	}
}

// Driver program to test main program functions
int main() {
	int h_a[N][N] = {
	  {  1,  0,  4,  0, 25,  0, 19,  0,  0, 10, 21,  8,  0, 14,  0,  6, 12,  9,  0,  0,  0,  0,  0,  0,  5},
	  {  5,  0, 19, 23, 24,  0, 22, 12,  0,  0, 16,  6,  0, 20,  0, 18,  0, 25, 14, 13, 10, 11,  0,  1, 15},
	  {  0,  0,  0,  0,  0,  0, 21,  5,  0, 20, 11, 10,  0,  1,  0,  4,  8, 24, 23, 15, 18,  0, 16, 22, 19},
	  {  0,  7, 21,  8, 18,  0,  0,  0, 11,  0,  5,  0,  0, 24,  0,  0,  0, 17, 22,  1,  9,  6, 25,  0,  0},
	  {  0, 13, 15,  0, 22, 14,  0, 18,  0, 16,  0,  0,  0,  4,  0,  0,  0, 19,  0,  0,  0, 24, 20, 21, 17},
	  { 12,  0, 11,  0,  6,  0,  0,  0,  0, 15,  0,  0,  0,  0, 21, 25, 19,  0,  4,  0, 22, 14,  0, 20,  0},
	  {  8,  0,  0, 21,  0, 16,  0,  0,  0,  2,  0,  3,  0,  0,  0,  0, 17, 23, 18, 22,  0,  0,  0, 24,  6},
	  {  4,  0, 14, 18,  7,  9,  0, 22, 21, 19,  0,  0,  0,  2,  0,  5,  0,  0,  0,  6, 16, 15,  0, 11, 12},
	  { 22,  0, 24,  0, 23,  0,  0, 11,  0,  7,  0,  0,  4,  0, 14,  0,  2, 12,  0,  8,  5, 19,  0, 25,  9},
	  { 20,  0,  0,  0,  5,  0,  0,  0,  0, 17,  9,  0, 12, 18,  0,  1,  0,  0,  7, 24,  0,  0,  0, 13,  4},
	  { 13,  0,  0,  5,  0,  2, 23, 14,  4, 18, 22,  0, 17,  0,  0, 20,  0,  1,  9, 21, 12,  0,  0,  8, 11},
	  { 14, 23,  0, 24,  0,  0,  0,  0,  0,  0,  0,  0, 20, 25,  0,  3,  4, 13,  0, 11, 21,  9,  5, 18, 22},
	  {  7,  0,  0, 11, 17, 20, 24,  0,  0,  0,  3,  4,  1, 12,  0,  0,  6, 14,  0,  5, 25, 13,  0,  0,  0},
	  {  0,  0, 16,  9,  0, 17, 11,  7, 10, 25,  0,  0,  0, 13,  6,  0,  0, 18,  0,  0, 19,  4,  0,  0, 20},
	  {  6, 15,  0, 19,  4, 13,  0,  0,  5,  0, 18, 11,  0,  0,  9,  8, 22, 16, 25, 10,  7,  0,  0,  0,  0},
	  {  0,  0,  0,  2,  0,  0, 10, 19,  3,  0,  1,  0, 22,  9,  4, 11, 15,  0, 20,  0,  0,  8, 23,  0, 25},
	  {  0, 24,  8, 13,  1,  0,  0,  4, 20,  0, 17, 14,  0,  0, 18,  0, 16, 22,  5,  0, 11,  0, 10,  0,  0},
	  { 23, 10,  0,  0,  0,  0,  0,  0, 18,  0,  6,  0, 16,  0,  0, 17,  1,  0, 13,  0,  0,  3, 19, 12,  0},
	  { 25,  5,  0, 14, 11,  0, 17,  0,  8, 24, 13,  0, 19, 23, 15,  9,  0,  0, 12,  0, 20,  0, 22,  0,  7},
	  {  0,  0, 17,  4,  0, 22, 15,  0, 23, 11, 12, 25,  0,  0,  0,  0, 18,  8,  0,  7,  0,  0, 14,  0, 13},
	  { 19,  6, 23, 22,  8,  0,  0,  1, 25,  4, 14,  2,  0,  3,  7, 13, 10, 11, 16,  0,  0,  0,  0,  0,  0},
	  {  0,  4,  0, 17,  0,  3,  0, 24,  0,  8, 20, 23, 11, 10, 25, 22,  0,  0,  0, 12, 13,  2, 18,  6,  0},
	  {  0,  0,  7, 16,  0,  0,  6, 17,  2, 21,  0, 18,  0,  0,  0, 19,  0,  0,  8,  0,  0,  0,  0,  4,  0},
	  { 18,  9, 25,  1,  2, 11,  0,  0, 13, 22,  4,  0, 21,  0,  5,  0, 23,  7,  0,  0, 15,  0,  3,  0,  8},
	  {  0, 21, 10,  0,  0, 12,  0, 20, 16,  0, 19,  0,  0,  0,  0, 15, 14,  4,  2, 18, 23, 25, 11,  7,  0}
	};
	int* d_a;      //Table
	hipMalloc((void**)&d_a, N * N * sizeof(int));
	// Copy Sudoku to device
	hipMemcpy(d_a, h_a, N * N * sizeof(int), hipMemcpyHostToDevice);
	dim3 dBlock(N, N);
	solve << <1, dBlock >> > (d_a);
	// Copy Sudoku back to host
	hipMemcpy(h_a, d_a, N * N * sizeof(int), hipMemcpyDeviceToHost);
	// Check if solved
	if (*h_a)
		print(h_a);
	else
		printf("No solution could be found.");
	hipFree(d_a);
	return 0;
}